#include "hip/hip_runtime.h"
///
/// matmultKernel00.cu
/// For CSU CS575 Spring 2011
/// Instructor: Wim Bohm
/// Based on code from the CUDA Programming Guide
/// Modified by Wim Bohm and David Newman
/// Created: 2011-01-27
/// Last Modified: 2011-02-23 DVN
///
/// Multiplies two matrices using CUDA: A x B = C
///
/// Copy this file and modify the MatMultKernel device function for
/// each of your experiments. 
///

#include "matmultKernel.h"

// Define a gpu kernel to perform matrix multiplication
// of A x B = C.
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){

  // matrix blocks
  float *Asub, *Bsub, *Csub;
  // Putting these into registers speeds access.
  int thread_row = threadIdx.y;
  int thread_col = threadIdx.x;
  int block_row = blockIdx.y;
  int block_col = blockIdx.x;

  // Each THREAD BLOCK computes one sub matrix Csub of C
  // EACH THREAD creates its own matrix descriptor Csub
  Csub = &C.elements[C.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * block_col];

  // Each thread computes one element of Csub in its copy of CValue
  float Cvalue00 = 0;
  float Cvalue01 = 0;
  float Cvalue10 = 0;
  float Cvalue11 = 0;
  
  // Loop over all sub matrices in block_row of A and block_col of B
  // required to compute Csub. Block multiply each pair of sub matrices
  // and accumulate results
  for (int m = 0;  m < (A.width / FOOTPRINT_SIZE); ++m){
    // Get Asub and Bsub descriptors
    Asub = &A.elements[A.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * m];
    Bsub = &B.elements[B.stride * FOOTPRINT_SIZE * m + FOOTPRINT_SIZE * block_col];

    // Copy ELEMENTS OF  ASub and Bsub into shared memory
    // EACH THREAD loads four ELEMENTs of ASub and four of Bsub

    // Notice: every thread declares shared_A and shared_B in shared memory
    //         even though a thread block has only one shared_A and one shared_B
    __shared__ float shared_A[FOOTPRINT_SIZE][FOOTPRINT_SIZE];
    __shared__ float shared_B[FOOTPRINT_SIZE][FOOTPRINT_SIZE];

    // Each thread copies four elements of shared_A and one element of shared_B

/*
    shared_A[thread_row][thread_col] = Asub[thread_row * A.stride + thread_col];
    shared_A[thread_row + BLOCK_SIZE][thread_col] = Asub[(thread_row + BLOCK_SIZE) * A.stride + thread_col];
    shared_A[thread_row][thread_col + BLOCK_SIZE] = Asub[thread_row * A.stride + thread_col + BLOCK_SIZE];
    shared_A[thread_row + BLOCK_SIZE][thread_col + BLOCK_SIZE] = Asub[(thread_row + BLOCK_SIZE) * A.stride + thread_col + BLOCK_SIZE];

    shared_B[thread_row][thread_col] = Bsub[thread_row * B.stride + thread_col];
    shared_B[thread_row][thread_col + BLOCK_SIZE] = Bsub[thread_row * B.stride + thread_col + BLOCK_SIZE];
    shared_B[thread_row + BLOCK_SIZE][thread_col] = Bsub[(thread_row + BLOCK_SIZE) * B.stride + thread_col];
    shared_B[thread_row + BLOCK_SIZE][thread_col + BLOCK_SIZE] = Bsub[(thread_row + BLOCK_SIZE) * B.stride + thread_col + BLOCK_SIZE];
*/

    int rh = thread_row % 2 ? 1 : 0;
#pragma unroll
    for (int i = 0; i < 4; ++i) {
        shared_A[thread_row / 2 + i * 8][thread_col + rh * BLOCK_SIZE] = Asub[(thread_row / 2 + i * 8) * A.stride + thread_col + rh * BLOCK_SIZE];
        shared_B[thread_row / 2 + i * 8][thread_col + rh * BLOCK_SIZE] = Bsub[(thread_row / 2 + i * 8) * B.stride + thread_col + rh * BLOCK_SIZE];
    }

    // Synchronize to ensure all shared values have been stored
    __syncthreads();

    // Do an inproduct of one row of shared_A and one col of shared_B
    // computing one Cvalue by accumulation
#pragma unroll
    for(int e=0; e<FOOTPRINT_SIZE; ++e) {
       Cvalue00 += shared_A[thread_row][e] * shared_B[e][thread_col];
       Cvalue01 += shared_A[thread_row][e] * shared_B[e][thread_col + BLOCK_SIZE];
       Cvalue10 += shared_A[thread_row + BLOCK_SIZE][e] * shared_B[e][thread_col];
       Cvalue11 += shared_A[thread_row + BLOCK_SIZE][e] * shared_B[e][thread_col + BLOCK_SIZE];
    }

    // Synchronize to ensure all Cvalues have been incremented
    // before reading in the next shared_A AND shared_B BLOCKS
    __syncthreads();
  }

  // Write Csub to GLOBAL memory.
  // Each thread writes its own cell value.
  Csub[thread_row * C.stride + thread_col] = Cvalue00;
  Csub[thread_row * C.stride + thread_col + BLOCK_SIZE] = Cvalue01;
  Csub[(thread_row + BLOCK_SIZE) * C.stride + thread_col] = Cvalue10;
  Csub[(thread_row + BLOCK_SIZE) * C.stride + thread_col + BLOCK_SIZE] = Cvalue11;
}

